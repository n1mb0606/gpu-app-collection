#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

#define THREADS_PER_BLOCK 1024
#define THREADS_PER_SM 2048
#define BLOCKS_NUM 160
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 1

#define CONFLICT_COUNT 1	// Must be between 1 to 16 

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *res, uint32_t ConflictCount) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	//register T s1 = data1[gid];
	//register T s2 = data2[gid];
	//register T result = 0;
    
	register int atomic_loc = 0; 
	if ((gid % 16) < ConflictCount)
		atomic_loc = 0;
	else
		atomic_loc = gid;

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
		atomicAdd(&data1[atomic_loc], 10);
	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	// write time and data back to memory
	startClk[gid] = start;
	stopClk[gid] = stop;
	res[gid] = data1[0];
}

int main(int argc, char ** argv){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	int32_t *data1 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *data2 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *res = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	int32_t *data1_g;
	//int32_t *data2_g;
	int32_t *res_g;

	//	Extract Cmdline Args
	uint32_t ConflictCount = 0;
	if (argc < 2) {
		printf("Usage : atomics_add_bw_profile [# Conflict Atomics]     \n");
		printf("        [# Diverged Atomics]  must be between 1 and 16  \n");
		return -1;
	}
	else {
		ConflictCount = atoi(argv[1]);
		printf(" Atomic : %d, Diverged %d \n", ConflictCount, 16 - ConflictCount);
	}

	for (uint32_t i = 0; i < TOTAL_THREADS; i++) {
		data1[i] = (int32_t)i;
		//data2[i] = (int32_t)i;
	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, TOTAL_THREADS*sizeof(int32_t)) );
	//gpuErrchk( cudaMalloc(&data2_g, TOTAL_THREADS*sizeof(int32_t)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(int32_t)) );

	gpuErrchk( hipMemcpy(data1_g, data1, TOTAL_THREADS*sizeof(int32_t), hipMemcpyHostToDevice) );
	//gpuErrchk( cudaMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(int32_t), cudaMemcpyHostToDevice) );

	max_flops<int32_t><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, res_g, ConflictCount);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(int32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(data2, data1_g, TOTAL_THREADS*sizeof(int32_t), hipMemcpyDeviceToHost) );
	printf("Found GPU Data Value = %d %d %d %d\n", data2[0], data2[1], data2[2], data2[3]);
	printf("Found GPU Result Value = %d %d %d %d\n", res[0], res[1], res[2], res[3]);

	float bw;
	uint32_t total_time = *std::max_element(&stopClk[0],&stopClk[TOTAL_THREADS-1])-*std::min_element(&startClk[0],&startClk[TOTAL_THREADS-1]);
	bw = ((float)(REPEAT_TIMES*TOTAL_THREADS*4)/(float)(total_time));
	printf("int32 bendwidth = %f (byte/clk)\n", bw);
	printf("Total Clk number = %u \n", total_time);

	return 0;
} 

